#include "hip/hip_runtime.h"
#include <stdio.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  include <windows.h>
#endif
#include "cuda_gl_interop.h"

#include "vtkBinningFluorescenceRendererCUDA.h"


#define BLOCKS 16
#define THREADS_PER_BLOCK 128

//////////////////////////////////////////////////////////////////////////////
// Splits an RGBA image into individual RGB channel buffers
//////////////////////////////////////////////////////////////////////////////
__global__ void splitPixelsKernel(float4 *pixels, int n,
                                  float *r, float *g, float *b) {
  const int tid     = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
  const int threadN = __mul24(blockDim.x, gridDim.x);

  // Naive version. Make use of shared memory in future
  // to exploit coalesced reads and writes.
  for (int i = tid; i < n; i += threadN) {
    float4 pixel = pixels[i];
    r[i] = pixel.x;
    g[i] = pixel.y;
    b[i] = pixel.z;
  }
}


extern "C"
hipError_t
splitPixels(float4 *pixels, int width, int height,
            float *r, float *g, float *b) {
  int n = width*height;
  dim3 grid(BLOCKS);
  dim3 block(THREADS_PER_BLOCK);

  splitPixelsKernel<<<grid, block>>>(pixels, n, r, g, b);

  return hipGetLastError();
}


//////////////////////////////////////////////////////////////////////////////
// Merges RGB channels into an RGBA image. Alpha value is 1.0.
//////////////////////////////////////////////////////////////////////////////
__global__ void mergePixelsKernel(float4 *pixels, int n,
                                  float *r, float *g, float *b) {

  const int tid     = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
  const int threadN = __mul24(blockDim.x, gridDim.x);

  // Naive version. Make use of shared memory in future
  // to exploit coalesced reads and writes.
  for (int i = tid; i < n; i += threadN) {
    pixels[i] = make_float4(r[i], g[i], b[i], 1.0f);

  }
}


extern "C"
hipError_t
mergePixels(float4 *pixels, int width, int height, float *r, float *g, float *b) {
  int n = width*height;
  dim3 grid(BLOCKS);
  dim3 block(THREADS_PER_BLOCK);

  mergePixelsKernel<<<grid, block>>>(pixels, n, r, g, b);

  return hipGetLastError();
}


//////////////////////////////////////////////////////////////////////////////
// Forward FFT
//////////////////////////////////////////////////////////////////////////////
extern "C"
void
forwardFFT(float *in, Complex *out, int width, int height, hipfftHandle plan) {
  CUFFT_SAFE_CALL(hipfftExecR2C(plan, (hipfftReal*) in, (hipfftComplex*) out));
}



//////////////////////////////////////////////////////////////////////////////
// Inverse FFT
//////////////////////////////////////////////////////////////////////////////
extern "C"
void
inverseFFT(Complex *in, float *out, int width, int height, hipfftHandle plan) {
  CUFFT_SAFE_CALL(hipfftExecC2R(plan, (hipfftComplex*) in, (hipfftReal*) out));
}


//////////////////////////////////////////////////////////////////////////////
// Does a component-wise multiply across two arrays of complex values.
//////////////////////////////////////////////////////////////////////////////
__global__ void complexMultiplyKernel(Complex *c1, Complex *c2, Complex *result, float scale, int n) {
  const int tid     = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
  const int threadN = __mul24(blockDim.x, gridDim.x);

  for (int i = tid; i < n; i += threadN) {
    result[i] = complexMulAndScale(c1[i], c2[i], scale);
  }
}


extern "C"
hipError_t
complexMultiply(Complex *c1, Complex *c2, Complex *result, int n, float scale) {
  dim3 grid(BLOCKS);
  dim3 block(THREADS_PER_BLOCK);

  complexMultiplyKernel<<<grid, block>>>(c1, c2, result, scale, n);

  return hipGetLastError();
}
